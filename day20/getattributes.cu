#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int device;
    hipGetDevice(&device);  // Get the current device

    // Query maximum number of threads per block
    int maxThreadsPerBlock;
    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, device);
    printf("Max threads per block: %d\n", maxThreadsPerBlock);

    // Query maximum number of blocks per grid (in each dimension)
    int maxGridDimX, maxGridDimY, maxGridDimZ;
    hipDeviceGetAttribute(&maxGridDimX, hipDeviceAttributeMaxGridDimX, device);
    hipDeviceGetAttribute(&maxGridDimY, hipDeviceAttributeMaxGridDimY, device);
    hipDeviceGetAttribute(&maxGridDimZ, hipDeviceAttributeMaxGridDimZ, device);
    printf("Max grid dimensions: X = %d, Y = %d, Z = %d\n", maxGridDimX, maxGridDimY, maxGridDimZ);

    // Query maximum block dimensions (in each dimension)
    int maxBlockDimX, maxBlockDimY, maxBlockDimZ;
    hipDeviceGetAttribute(&maxBlockDimX, hipDeviceAttributeMaxBlockDimX, device);
    hipDeviceGetAttribute(&maxBlockDimY, hipDeviceAttributeMaxBlockDimY, device);
    hipDeviceGetAttribute(&maxBlockDimZ, hipDeviceAttributeMaxBlockDimZ, device);
    printf("Max block dimensions: X = %d, Y = %d, Z = %d\n", maxBlockDimX, maxBlockDimY, maxBlockDimZ);


    // Query the maximum shared memory per block
    int maxSharedMemoryPerBlock;
    hipDeviceGetAttribute(&maxSharedMemoryPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, device);
    printf("Max shared memory per block: %d KB\n", maxSharedMemoryPerBlock/(1024));

    // Query the maximum shared memory per multiprocessor (SM)
    int maxSharedMemoryPerMultiprocessor;
    hipDeviceGetAttribute(&maxSharedMemoryPerMultiprocessor, hipDeviceAttributeMaxSharedMemoryPerMultiprocessor, device);
    printf("Max shared memory per multiprocessor: %d KB\n", maxSharedMemoryPerMultiprocessor/(1024));

    // Query the maximum number of registers per block
    int maxRegistersPerBlock;
    hipDeviceGetAttribute(&maxRegistersPerBlock, hipDeviceAttributeMaxRegistersPerBlock, device);
    printf("Max registers per block: %d\n", maxRegistersPerBlock);

    // Query the maximum number of registers per multiprocessor
    int maxRegistersPerMultiprocessor;
    hipDeviceGetAttribute(&maxRegistersPerMultiprocessor, hipDeviceAttributeMaxRegistersPerMultiprocessor, device);
    printf("Max registers per multiprocessor: %d\n", maxRegistersPerMultiprocessor);


    return 0;
}
