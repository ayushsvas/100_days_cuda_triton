#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <torch/extension.h>

__global__ void transpose(float*a, float*b, int col_size, int row_size){
        int tidx = threadIdx.x + threadIdx.y * blockDim.x;
        int tidy = threadIdx.y + threadIdx.x * blockDim.y;
       
        if (tidx>=col_size*row_size || tidy >= col_size*row_size) return;
       
        b[tidx] = a[tidy];

}


int main(){
        const int col_size = 4;
        float arr[][col_size] = {{1.0,2.0,3.1,4.1}, {1.0,2.4,3.5,4.7}, {1.2,2.2,3.2,4.2}, {1.1,2.1,3.1,4.1}};

        const int row_size = sizeof(arr) / (col_size*sizeof(float));
        std:: cout<<row_size<<" "<<col_size<<"\n";
        
        float *da, *db;
        hipMalloc(&da, col_size*row_size*sizeof(float));
        hipMalloc(&db, col_size*row_size*sizeof(float));
    
        hipMemcpy(da, arr, col_size*row_size*sizeof(float), hipMemcpyHostToDevice);

        dim3 numThreadsPerBlock(col_size,row_size);
    
        transpose <<< 1, numThreadsPerBlock >>> (da,db,col_size,row_size);

        hipMemcpy(arr, db, col_size*row_size*sizeof(float), hipMemcpyDeviceToHost);

        for (int i=0;i<col_size;++i){
            for (int j=0;j<row_size;++j){
                std:: cout<<arr[i][j]<<" "; 
            }
            std:: cout<<"\n";
        }   

        hipFree(da);
        hipFree(db);

        return 0;



}

