#include <hip/hip_runtime.h>

# define ERROR_CHECK(call){\
    hipError_t err = call; \
    if (err != hipSuccess){ \
    printf("%s in %s at line %d \n", \
        hipGetErrorString(err), __FILE__, __LINE__);\
    exit(EXIT_FAILURE);\
    }\
} 


// Error checking macro
#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in line %d: %s\n", __LINE__, \
                hipGetErrorString(err)); \
        return err; \
    } \
}

#define CHECK_CUFFT(call) { \
    hipfftResult_t err = call; \
    if (err != HIPFFT_SUCCESS) { \
        fprintf(stderr, "CUFFT error in line %d: %d\n", __LINE__, err); \
        return hipErrorUnknown; \
    } \
}

#define CHECK_CUBLAS(call)                                                      \
    do {                                                                        \
        hipblasStatus_t status = call;                                           \
        if (status != HIPBLAS_STATUS_SUCCESS) {                                  \
            std::cerr << "cuBLAS error\n";                                      \
            exit(EXIT_FAILURE);                                                 \
        }                                                                       \
    } while (0)
