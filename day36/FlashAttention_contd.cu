#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>




constexpr int BLOCK_SIZE = 16;
constexpr int HIDDEN_DIM = 128;

extern "C" __global__ void FlashAttention(
    float* output,
    float* output_lse,
    const float* query,
    const float* key,
    const float* value,
    const float scale,
    const int N_out,
    const int N_inp
) {
    __shared__ float q_block[BLOCK_SIZE][HIDDEN_DIM];
    __shared__ float k_block[BLOCK_SIZE][HIDDEN_DIM];
    __shared__ float v_block[BLOCK_SIZE][HIDDEN_DIM];
    
    const int tx = threadIdx.x; // these are local to a block
    const int ty = threadIdx.y;
    const int row = blockIdx.x * BLOCK_SIZE + tx; // getting global index
    
    // these will be stored in registers. Every thread with its own copy
    float m_i = -INFINITY;
    float l_i = 0.0f;
    float o_i[HIDDEN_DIM] = {0.0f};
    
    // Below we traverse through the query block in blocks of size BLOCK_SIZExHIDDEN_DIM
    // This assigning happens in parallel (hopefully) for all the threads in BLOCK_SIZExHIDDEN_DIM
    // Only 1 tile (tile == 0) in the shared memory at a time.
    for (int tile = 0; tile < (N_inp + BLOCK_SIZE - 1)/BLOCK_SIZE; ++tile) {
        if (tile == 0 && row < N_out) {
            for (int d = 0; d < HIDDEN_DIM; d += blockDim.y) {
                int d_idx = d + ty;
                if (d_idx < HIDDEN_DIM) {
                    q_block[tx][d_idx] = query[row*HIDDEN_DIM+d_idx]; // in memory things are flattened
                }
            }
        }
        __syncthreads();
        
        if (tile * BLOCK_SIZE + ty < N_inp && row < N_out) {
            for (int d = 0; d < HIDDEN_DIM; d += blockDim.y) {
                int d_idx = d + ty;
                if (d_idx < HIDDEN_DIM) {
                    k_block[tx][d_idx] = key[(tile * BLOCK_SIZE+tx)*HIDDEN_DIM+d_idx];
                    v_block[tx][d_idx] = value[(tile * BLOCK_SIZE+tx)*HIDDEN_DIM+d_idx];
                }
            }
        }
        __syncthreads();
        