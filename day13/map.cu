#include <stdio.h> 
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void map_fn(int *a, int *b, int n){
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tidx < n){
        b[tidx] = a[tidx] + 10;
    }
}

int main(){
    int a[10];
    int o[10];
    printf("Enter 10 digits...\n");
    for (int i=0;i<10;++i){
        scanf("%d", &a[i]); 
    }

    int *g_a;
    int *g_o;

    hipMalloc((void**)&g_a, 10*sizeof(int));
    hipMalloc((void**)&g_o, 10*sizeof(int));

    hipMemcpy(g_a, a, 10*sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 10;
    int blocksPerGrid = 10;

    map_fn <<< blocksPerGrid,threadsPerBlock >>> (g_a, g_o, 10);

    hipMemcpy(o, g_o, 10*sizeof(int), hipMemcpyDeviceToHost);

    printf("After adding...\n");
    for (int i=0;i<10;++i){
        printf("%d\n", o[i]);
    }

    hipFree(g_a);
    hipFree(g_o);

    return 0;

    
}